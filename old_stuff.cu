#include "hip/hip_runtime.h"
#include "util.cuh"

// we only do square size (easier to manage)

// optimized for large area
DEVICEABLE int get_biome_area(int64_t world_seed, int32_t x, int32_t z, uint8_t size) {
    uint32_t cache_size = get_max_size(size);
    // technically I need 16 values, I might make it like so
    auto *cache1 = new uint8_t[cache_size];
    auto *cache2 = new uint8_t[cache_size];
    int64_t layerSeed, localSeed;
    // Continent layer
    layerSeed = get_layer_seed_precomputed(world_seed, SALT_1);
    printf("%lld %lld\n", layerSeed, SALT_1);

    return 0;
}



// optimized for single point
DEVICEABLE int get_biome(int64_t world_seed, int32_t pos_x, int32_t pos_z, int32_t width, int32_t height) {
    int32_t * layer_sizes= get_size(width,height);
    int32_t * layer_positions= get_pos(pos_x,pos_z);
    int64_t layer_seed, local_seed;
    int32_t current_x, current_z;
    int32_t size_x, size_z;
    int32_t * coords,*sizes;
    uint32_t cache_size = get_max_size(1);
    auto *cache1 = new uint8_t[cache_size];
    auto *cache2 = new uint8_t[cache_size];
    // Continent layer
    layer_seed = get_layer_seed_precomputed(world_seed, SALT_1);
    coords= get_for_layer(layer_positions,0);
    print_array_i32(coords,4);
    current_x = min(coords[0],coords[2]);
    current_z = min(coords[1],coords[3]);
    sizes= get_for_layer(layer_sizes,0);
    size_x= abs(coords[0]-coords[2])+ max(sizes[0],sizes[2]);
    size_z= abs(coords[1]-coords[3])+ max(sizes[1],sizes[3]);
    printf("%d %d %d %d\n", current_x, current_z, size_x,size_z);
    for (int32_t z = 0; z < size_z; z++) {
        for (int32_t x = 0; x < size_x; x++) {
            local_seed = get_local_seed(layer_seed, current_x + x, current_z + z);
            cache1[x + z * size_x] = next_int_without(local_seed, 10) != 0 ? OCEAN : PLAINS;
        }
    }
    // make spawn a bit more liveable
    if (-size_x < current_x && current_x <= 0 && -size_z < current_z && current_z <= 0) {
        cache1[-current_x + -current_z * size_x] = 1;
    }
    print_array_u8(cache1, cache_size);
    // scale layer
    layer_seed = get_layer_seed_precomputed(world_seed, SALT_2000);
    coords= get_for_layer(layer_positions,1);
    print_array_i32(coords,4);
    current_x = min(coords[0],coords[2]);
    current_z = min(coords[1],coords[3]);
    sizes= get_for_layer(layer_sizes,1);
    size_x= abs(coords[0]-coords[2])+ max(sizes[0],sizes[2]);
    size_z= abs(coords[1]-coords[3])+ max(sizes[1],sizes[3]);
    printf("%d %d %d %d\n", current_x, current_z, size_x,size_z);

    return 0;
}


DEVICEABLE int get_single_biome(int64_t world_seed, int32_t pos_x, int32_t pos_z) {
    return get_biome(world_seed, pos_x, pos_z, 1, 1);
}

__global__  void biome_gen(uint64_t world_seed) {
    printf("%lld %lld\n", mix_salt(1), get_layer_seed(1, 1));
}


void biome_gen_d(uint64_t world_seed) {
    printf("%lld %lld\n", mix_salt(1), get_layer_seed(1, 1));
}


int main() {
    uint64_t seed = 1LL;
    //biome_gen<<<1, 1>>>(seed);
    get_biome(1, 7973,0,10000,10000);
    return 0;
}
