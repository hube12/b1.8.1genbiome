#include "hip/hip_runtime.h"
#include "util.cuh"

// we only do square size (easier to manage)

// optimized for large area
DEVICEABLE int get_biome_area(int64_t world_seed, int32_t x, int32_t z, uint8_t size) {
    uint32_t cache_size = get_max_size(size);
    // technically I need 16 values, I might make it like so
    auto *cache1 = new uint8_t[cache_size];
    auto *cache2 = new uint8_t[cache_size];
    int64_t layerSeed, localSeed;
    // Continent layer
    layerSeed = get_layer_seed_precomputed(world_seed, SALT_1);
    printf("%lld %lld\n", layerSeed, SALT_1);

    return 0;
}

DEVICEABLE void print_array(uint8_t *arr, uint32_t size) {
    for (uint32_t i = 0; i < size; i++)
        printf("%d,", arr[i]);
    printf("\n");
}

// optimized for single point
DEVICEABLE int get_biome(int64_t world_seed, int32_t pos_x, int32_t pos_z, uint8_t start_layer_size, uint32_t start_scale) {
    int32_t * layer_sizes= get_size(1,1);
    int32_t * layer_positions= get_pos(pos_x,pos_z);
    uint32_t scale = start_scale;
    int64_t layer_seed, local_seed;
    int32_t current_x, current_z;
    int32_t * coords;
    uint32_t cache_size = get_max_size(1);
    auto *cache1 = new uint8_t[cache_size];
    auto *cache2 = new uint8_t[cache_size];
    // Continent layer
    layer_seed = get_layer_seed_precomputed(world_seed, SALT_1);
    coords= get_for_layer(layer_positions,0);
    current_x = layer_sizes[2+]
    current_z = (pos_z - 7974) / (int32_t) scale;
    printf("%d %d %d\n", current_x, current_z, layer_size);
    for (uint8_t z = 0; z < layer_size; z++) {
        for (uint8_t x = 0; x < layer_size; x++) {
            local_seed = get_local_seed(layer_seed, current_x + x, current_z + z);
            cache1[x + z * layer_size] = next_int_without(local_seed, 10) != 0 ? OCEAN : PLAINS;
        }
    }
    // make spawn a bit more liveable
    if (-layer_size < current_x && current_x <= 0 && -layer_size < current_z && current_z <= 0) {
        cache1[-current_x + -current_z * layer_size] = 1;
    }
    print_array(cache1, cache_size);
    return 0;
}


DEVICEABLE int get_single_biome(int64_t world_seed, int32_t pos_x, int32_t pos_z) {
    return get_biome(world_seed, pos_x, pos_z, 7, 8192);
}

__global__  void biome_gen(uint64_t world_seed) {
    printf("%lld %lld\n", mix_salt(1), get_layer_seed(1, 1));
}


void biome_gen_d(uint64_t world_seed) {
    printf("%lld %lld\n", mix_salt(1), get_layer_seed(1, 1));
}


int main() {
    uint64_t seed = 1LL;
    //biome_gen<<<1, 1>>>(seed);
    get_single_biome(1, 0, 0);
    return 0;
}
